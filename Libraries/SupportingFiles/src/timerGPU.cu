#include "timerGPU.hpp"

#include <iostream>
#include <iomanip>
#include <string>

#include <hip/hip_runtime.h>


GpuTimer::GpuTimer() {
    hipEventCreate(&clock_start);
    hipEventCreate(&clock_stop);
}

GpuTimer::~GpuTimer() {
    hipEventDestroy(clock_start);
    hipEventDestroy(clock_stop);
}

void GpuTimer::start() {
    hipEventRecord(clock_start, 0);
}

void GpuTimer::stop() {
    hipEventRecord(clock_stop, 0);
}

float GpuTimer::elapsed() {
    float elapsed;
    hipEventSynchronize(clock_stop);
    hipEventElapsedTime(&elapsed, clock_start, clock_stop);
    return elapsed;
}

void GpuTimer::printElapsed(
    std::string timed_fct_name
) {
    float elapsed = this->elapsed();
    std::cout << "Executing " << timed_fct_name << " took " << std::setw(8) << elapsed << " ms. ";
}
