#include "cudaMemory.hpp"

#include <hip/hip_runtime.h>
#include "checkCudaErrors.hpp"


void allocCudaMem(
    void** d_DataPtrPtr,
    const unsigned int size
) {
    checkCudaErrors(hipMalloc(d_DataPtrPtr, size));
    checkCudaErrors(hipMemset(*d_DataPtrPtr, 0, size));
}

void gpuMemFree(
    void** d_DataPtrPtr
) {
    checkCudaErrors(hipFree(*d_DataPtrPtr));
    *d_DataPtrPtr = nullptr;
}

void memsetZero(
    void* d_DataPtr,
    const unsigned int size
) {
    checkCudaErrors(hipMemset(d_DataPtr, 0, size));
}

void memcpyCPUtoGPU(
    void* h_DataPtr,
    void* d_DataPtr,
    const unsigned int size
) {
    checkCudaErrors(hipMemcpy(d_DataPtr, h_DataPtr, size, hipMemcpyHostToDevice));
}

void memcpyGPUtoCPU(
    void* d_DataPtr,
    void* h_DataPtr,
    const unsigned int size
) {
    checkCudaErrors(hipMemcpy(h_DataPtr, d_DataPtr, size, hipMemcpyDeviceToHost));
}

void memcpyGPUtoGPU(
    void* d_DataFromPtr,
    void* d_DataToPtr,
    const unsigned int size
) {
    checkCudaErrors(hipMemcpy(d_DataToPtr, d_DataFromPtr, size, hipMemcpyDeviceToDevice));
}
